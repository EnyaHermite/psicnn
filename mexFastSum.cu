#include "hip/hip_runtime.h"
#include "mex.h"
#include "matrix.h"
#include "gpu/mxGPUArray.h"
#include <hip/hip_runtime.h>
#include <>
#include <hipblas.h>
#include <cassert>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>

#define VL_CUDA_NUM_THREADS 1024
#define maxStreams 16 // maximum number of streams used for concurrency

using namespace std;

inline size_t divideAndRoundUp(size_t a, size_t b)
{
	return (a + b - 1) / b;
}

__global__ void
setToOnes(float *data, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x; // 1D grid of 1D blocks
	if (index < size) data[index] = 1;
}

void createStreams(hipStream_t *streams, int N)
{
	for (int i = 0; i < N; i++)
	{
		hipStreamCreate(&(streams[i]));
	}
}

void destroyStreams(hipStream_t *streams, int N)
{
	for (int i = 0; i < N; i++)
	{
		hipStreamDestroy(streams[i]);
	}
}


bool speedup_sum(float *output,
	float const *input,
	unsigned int const *nodeSize,
	unsigned int const Nin,
	unsigned int const Nout,
	unsigned int const Fin)
{
	assert(output);
	assert(input);

	hipError_t hipError_t;
	hipblasStatus_t cublasError;
	hipblasHandle_t handle;
	hipStream_t *streams = NULL;

	bool status = true;
	hipblasCreate(&handle);
	streams = (hipStream_t *)malloc(maxStreams * sizeof(hipStream_t));
	createStreams(streams, maxStreams);

	float *allOnesMemory = NULL;
	hipError_t = hipMalloc((void **)&allOnesMemory, Nin * sizeof(float));
	if (hipError_t != hipSuccess) { goto done; }
	setToOnes << < divideAndRoundUp(Nin, (size_t)VL_CUDA_NUM_THREADS), VL_CUDA_NUM_THREADS, 0, streams[0] >> >
		((float *)allOnesMemory, Nin);

	hipError_t = hipDeviceSynchronize();
	if (hipError_t != hipSuccess) { goto done; }

	// parent-wise memcpy or summation
	float  alpha, beta;
	alpha = 1;
	beta = 0;
	int count;
	count = 0;
	for (int iter = 0; iter < Nout; ++iter)
	{
		unsigned int convSize = nodeSize[iter];
		ptrdiff_t inputOffset = Fin * count;
		ptrdiff_t outputOffset = Fin * iter;

		if (convSize == 1) // copy if for-loop end with convSize=1 nodes
		{
			hipError_t = hipMemcpyAsync((float *)output + outputOffset,
				(float const *)input + inputOffset,
				Fin * sizeof(float), hipMemcpyDeviceToDevice, streams[iter % maxStreams]);
			if (hipError_t != hipSuccess) { goto done; }
		}
		else
		{
			hipblasSetStream(handle, streams[iter % maxStreams]);
			cublasError = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
				Fin, 1, convSize,
				&alpha,
				(float const *)input + inputOffset, Fin,
				(float const *)allOnesMemory, 1,
				&beta,
				(float *)output + outputOffset, Fin);
			if (cublasError != HIPBLAS_STATUS_SUCCESS) { goto done; }
		}

		count += convSize;
	}

	hipError_t = hipDeviceSynchronize();
	if (hipError_t != hipSuccess) { goto done; }

done:
	if (hipError_t != hipSuccess || cublasError != HIPBLAS_STATUS_SUCCESS) { status = false; }
	if (allOnesMemory) hipFree(allOnesMemory);
	if (streams) destroyStreams(streams, maxStreams);
	hipblasDestroy(handle);

	return status;
}


void mexFunction(int nlhs, mxArray *plhs[], int nrhs,
	mxArray const *prhs[])
{
	float const  *input;
	float *output;
	unsigned int Fin, Nin, Nout;
	unsigned int const *nodeSize;
	bool err = false;
	char *errMsg;

	mxGPUArray const *dev_input;
	mxGPUArray *dev_output;
	mwSize const *dims;
	mwSize dims_4D[] = { 1, 1, 1, 1 };
	mwSize ndim;

	mxInitGPU(); /* Initialize the MathWorks GPU API. */

				 /* -------------------------------------------------------------- */
				 /*										Check the input arguments */
				 /* -------------------------------------------------------------- */

	if (nrhs < 2) { mexErrMsgTxt("Not enough input arguments."); }
	if (nrhs > 2) { mexErrMsgTxt("Too many inputs."); }

	// get pointer to gpuArray input
	dev_input = mxGPUCreateFromMxArray(prhs[0]);
	if (mxGPUGetClassID(dev_input) == mxSINGLE_CLASS)
	{
		input = (float const *)mxGPUGetDataReadOnly(dev_input);
	}
	else
	{
		mexErrMsgTxt("Only single format of network input is supported.");
	}

	// get pointer to cpuArray map
	if (mxGetClassID(prhs[1]) == mxUINT32_CLASS)
	{
		nodeSize = (unsigned int const *)mxGetData(prhs[1]);
		Nout = mxGetNumberOfElements(prhs[1]);
	}
	else
	{
		mexErrMsgTxt("Only unsigned int format of nodeSize is supported.");
	}

	// parse the network input
	ndim = mxGPUGetNumberOfDimensions(dev_input);
	dims = mxGPUGetDimensions(dev_input);
	if (ndim<3 || ndim>4)
	{
		mexErrMsgTxt("The network input must be a 4D matrix.");
	}
	else
	{
		Fin = dims[0] * dims[1] * dims[2];
		Nin = (ndim == 3) ? 1 : dims[3];
	}

	/* -------------------------------------------------------------- */
	/*													   Do the work*/
	/* -------------------------------------------------------------- */
	dims_4D[2] = Fin; dims_4D[3] = Nout;
	dev_output = mxGPUCreateGPUArray(4, dims_4D, mxSINGLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	output = (float *)mxGPUGetData(dev_output);
	bool status = speedup_sum(output, input, nodeSize, Nin, Nout, Fin);

	if (status)
	{
		plhs[0] = mxGPUCreateMxArrayOnGPU(dev_output);
	}
	else
	{
		err = true;
		errMsg = "The speed-up summation is failed.";
	}

	mxGPUDestroyGPUArray(dev_input);
	mxGPUDestroyGPUArray(dev_output);
	if (err) mexErrMsgTxt(errMsg);
}

